#include "hip/hip_runtime.h"
/*********************************
Developer: Maksudul Alam
Oak Ridge National Laboratory
*********************************/

#include "cuPPA.cuh"
#include <hip/hip_runtime_api.h>

void cuPPA::generateGraph(ExperimentConfiguration conf)
{
    hipDeviceReset();

    _N = conf._n1;
    _M = conf._x1;
    _p = conf._alpha;

#if WAITING_QUEUE_IN_GLOBAL_MEMORY
    const int WQ_MAX_SIZE = 1024;
#else
    const int WQ_MAX_SIZE = 48;
#endif
    const int nBlocks = conf._blockSize;
    const int MAX_THREADS_PER_BLOCK = conf._threadsPerBlock;
    const int MAX_QUEUE_CAPACITY = WQ_MAX_SIZE * 1024 / sizeof(EdgeType);

    //INITIAL SETTINGS
    int nThreads = conf._threadsPerBlock;
    int CURRENT_QUEUE_SIZE = MAX_QUEUE_CAPACITY / nThreads;
    int VERTICES_PER_THREAD = MAX_QUEUE_CAPACITY / nThreads / _M;
    int REQUIRED_QUEUE = _M * VERTICES_PER_THREAD;
    const int MIN_WARP_SIZE = nThreads * nBlocks * conf._warpSize;
    const int MAX_BLOCK_SIZE = ceil((double) (_N - _M - 1) / (double) conf._stages / MIN_WARP_SIZE) * MIN_WARP_SIZE; //BlockSize is multiple of MIN_WARP_SIZE
    int stepSize = ceil((double) (_N - _M - 1) / MAX_BLOCK_SIZE);

#if USE_DEFAULT_CURAND
    hiprandState *devStates;
    checkCudaErrors(hipMalloc((void ** ) &devStates, nBlocks * nThreads * sizeof(hiprandState)));
    //    unsigned long long seed = 1234;
    unsigned long long seed = std::time(0);
    setup_curand_kernel<<<nBlocks, nThreads>>>(devStates, seed); //Creates nBlocks * nThreads Streams!

    {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            printf("%d: Error 0:  %s\n", __LINE__, hipGetErrorString(err));
    }
#else
    hiprandStateMRG32k3a *devStates;
    checkCudaErrors(hipMalloc((void ** ) &devStates, nBlocks * nThreads * sizeof(hiprandStateMRG32k3a)));
    unsigned long long seed = 1234;
    setup_curand_kernel_MRG32k3a<<<nBlocks, nThreads>>>(devStates, seed);    //Creates nBlocks * nThreads Streams!
#endif

    checkCudaErrors(hipMalloc((void** ) &_dGraph, _N * _M * sizeof(NodeType))); //Whole Graph Size

    cuBSTVertex *F;
    checkCudaErrors(hipMalloc((void** ) &F, MAX_BLOCK_SIZE * sizeof(cuBSTVertex))); //cuBSTVertex

    EdgeIndexType * children;
    checkCudaErrors(hipMalloc((void** ) &children, 2 * MAX_BLOCK_SIZE * _M * sizeof(EdgeIndexType))); //Left and Right Children

    int *dMaxQueueSizes;
    int hMaxQueueSize = 0;

#if WAITING_QUEUE_IN_GLOBAL_MEMORY
    EdgeType *dQ; //Shared Memory Size
    checkCudaErrors(hipMalloc((void** ) &dQ, WQ_MAX_SIZE * 1024 * nBlocks));//Waiting Queue
    checkCudaErrors(hipMalloc((void** ) &dMaxQueueSizes, nThreads * nBlocks * sizeof(int)));
    hipMemset(dMaxQueueSizes, 0, nThreads * nBlocks * sizeof(int));

    int *hMaxQueueSizes = (int*) malloc(nThreads * nBlocks * sizeof(int));
    memset(hMaxQueueSizes, 0, nThreads * nBlocks * sizeof(int));

#else
    checkCudaErrors(hipMalloc((void** ) &dMaxQueueSizes, 32 * sizeof(int)));
    hipMemset(dMaxQueueSizes, 0, 32 * sizeof(int));
#endif

#if SHOW_QUEUE_OUTPUT
    std::ofstream maxQFile("Q-Original.csv");
    maxQFile << "Round,MaxQ\n";
#endif

#if SHOW_INTERNAL_TIMINGS
    std::ofstream stepwiseTimingFile("StepwiseTiming-Original.csv");
    stepwiseTimingFile << "Round,nBlocks,nThreads,Edges,MaxQ,Initialization Time,Computation Time\n";
#endif

    printf("--------------------------------------------------------------------------------------------------------------------------------------------\n");
    cout << "BLOCKS             : \t" << nBlocks << "\t| " << "MAX_THREADS_PER_BLOCK: \t" << nThreads << "\t| " << "nThreads     : " << nThreads << endl;
    printf("--------------------------------------------------------------------------------------------------------------------------------------------\n");
    cout << "MAX_QUEUE_CAPACITY : \t" << MAX_QUEUE_CAPACITY << "\t| " << "REQUIRED_QUEUE       : \t" << REQUIRED_QUEUE << "\t| CURRENT_QUEUE: " << MAX_QUEUE_CAPACITY / nThreads << endl;
    printf("--------------------------------------------------------------------------------------------------------------------------------------------\n");
    cout << "VERTICES_PER_THREAD: \t" << VERTICES_PER_THREAD << "\t| " << "MAX_BLOCK_SIZE       : \t" << MAX_BLOCK_SIZE << "\t| " << "STEPS        : \t" << stepSize << endl;
    printf("--------------------------------------------------------------------------------------------------------------------------------------------\n");
    cout << "Random: \t" << (nBlocks * MAX_THREADS_PER_BLOCK * sizeof(hiprandState) / 1024.0 / 1024.0) << "MB" << "\t| " //
            << "Graph: \t" << (_N * _M * sizeof(NodeType) / 1024.0 / 1024.0) << "MB" << "\t| " //
            << "BST: \t" << (MAX_BLOCK_SIZE * sizeof(cuBSTVertex) / 1024.0 / 1024.0) << "MB" << "\t| " //
            << "Children: \t" << (2 * MAX_BLOCK_SIZE * _M * sizeof(EdgeIndexType) / 1024.0 / 1024.0) << "MB" << "\t| " //
            << endl;
    printf("--------------------------------------------------------------------------------------------------------------------------------------------\n");

    Timer t1;
    t1.start();
    initial_graph_rrp<<<nBlocks, nThreads>>>(_M, _dGraph);
    {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            printf("%d: Error 1:  %s\n", __LINE__, hipGetErrorString(err));
    }

    for (int iteration_index = 0; iteration_index < stepSize; iteration_index++)
    {
        NodeType u = _M + 1 + iteration_index * MAX_BLOCK_SIZE;
        NodeType v = u + MAX_BLOCK_SIZE - 1;
        v = v > _N - 1 ? _N - 1 : v;
        NodeType B = v - u + 1;

#if SHOW_INTERNAL_TIMINGS & 0
        if (iteration_index < conf._N_STAGE_TIMINGS)
        {
            printf("%d\tProcessing: %lld\t%lld\tTotal: %lld\n", iteration_index, u, v, B);
        }
#endif
        if (u < _N)
        {
#if SHOW_INTERNAL_TIMINGS
            Timer tVS;
            tVS.start();
#endif

#if 0
            //NOT SUPER FAST as expected
            //Vertex Setup can be done super fast without any dependency!
            int nThreadsVertexSetup = 512;
            int nBlocksVertexSetup = (B + nThreadsVertexSetup - 1) / nThreadsVertexSetup;
            initialize_vertex_structures_rrp_2<<<nBlocksVertexSetup, nThreadsVertexSetup>>>(u, B, M, _dGraph, children, F);
#else
#if 0
            //NOT FASTER, but expected it to be.
            hipMemset(children, 0, 2 * B * _M * sizeof(EdgeIndexType));
            hipMemset(_dGraph + (u * M), 0, B * _M * sizeof(NodeType));
            initialize_vertex_structures_rrp_fast<<<nBlocks, nThreads>>>(u, B, _M, _dGraph, children, F);
#else
            //Currently the fastest implementation
            initialize_vertex_structures_rrp<<<nBlocks, nThreads>>>(u, B, _M, _dGraph, children, F);
            {
                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                    printf("%d: Error 2:  %s\n", __LINE__, hipGetErrorString(err));
            }
#endif
#endif

#if SHOW_INTERNAL_TIMINGS
            hipDeviceSynchronize();
            tVS.stop();
#endif

#if SHOW_INTERNAL_TIMINGS
            Timer tGn;
            tGn.start();
#endif

#if WAITING_QUEUE_IN_GLOBAL_MEMORY
            generate_graph_rrp_by_batch_execute_copy_model<<<nBlocks, nThreads>>>(u, B, _M, _p, F, _dGraph, devStates, dQ, dMaxQueueSizes);
            {
                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                printf("%d: Error 3:  %s\n", __LINE__, hipGetErrorString(err));
            }
            hipMemcpy(hMaxQueueSizes, dMaxQueueSizes, nThreads * nBlocks * sizeof(int), hipMemcpyDeviceToHost);

            hipDeviceSynchronize();

            generate_graph_rrp_by_batch_resolve_queue<<<nBlocks, nThreads>>>(u, B, _M, _p, F, _dGraph, devStates, dQ, dMaxQueueSizes);
            {
                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                printf("%d: Error 4:  %s\n", __LINE__, hipGetErrorString(err));
            }
            hMaxQueueSize = 0;
            for (int ijk = 0; ijk < nThreads * nBlocks; ijk++)
            {
                if (hMaxQueueSize < hMaxQueueSizes[ijk])
                hMaxQueueSize = hMaxQueueSizes[ijk];
            }
#else
            generate_graph_rrp_by_batch<<<nBlocks, nThreads>>>(u, B, _M, _p, F, _dGraph, devStates, dMaxQueueSizes);
            {
                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                    printf("%d: Error 5:  %s\n", __LINE__, hipGetErrorString(err));
            }
            find_max_queue_size<<<1, 32>>>(dMaxQueueSizes);
            {
                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                    printf("%d: Error 6:  %s\n", __LINE__, hipGetErrorString(err));
            }
#endif

#if SHOW_INTERNAL_TIMINGS
            hipDeviceSynchronize();
            tGn.stop();
#endif

#if SHOW_QUEUE_OUTPUT
            if (iteration_index < conf._N_STAGE_TIMINGS)
            {
                int Cap = WQ_MAX_SIZE * 1024 / nThreads / sizeof(EdgeType);
                printf("************************************************************************************                      MAX:\t%d/%d\n", hMaxQueueSize, Cap);
                maxQFile << (iteration_index + 1) << "," << hMaxQueueSize << endl;
            }
#endif

#if SHOW_INTERNAL_TIMINGS
            if (iteration_index < conf._N_STAGE_TIMINGS)
            {
                printf("Step:\t%d\tVertex Setup Time:\t%f (%3.2f Mps)\t\tGeneration Time:\t%f (%3.2f Mps)\n", iteration_index, tVS.getElapsedTimeInMilliSec(), B / tVS.getElapsedTimeInMilliSec() / 1024.0, tGn.getElapsedTimeInMilliSec(),
                        B / tGn.getElapsedTimeInMilliSec() / 1024.0);
                stepwiseTimingFile << (iteration_index + 1) << "," << nBlocks << "," << nThreads << "," << B * _M << "," << hMaxQueueSize << "," << tVS.getElapsedTimeInSec() << "," << tGn.getElapsedTimeInSec() << endl;
            }
#endif
        }
    }

#if SHOW_QUEUE_OUTPUT
    maxQFile.close();
#endif

#if SHOW_INTERNAL_TIMINGS
    stepwiseTimingFile.close();
#endif

    hipDeviceSynchronize();
    t1.stop();

    printf("Computation Time: %f\n", t1.getElapsedTimeInSec());
    printf("\n\n");

    if (conf._FLAG_SHOW_DEGREE_DISTRIBUTION)
    {
        printDegreeDistribution(conf);
    }

    hipFree(devStates);
    hipFree(_dGraph);
    hipFree(children);
    hipFree(F);

    std::cout << "Ended" << std::endl;
    hipDeviceSynchronize();
}
